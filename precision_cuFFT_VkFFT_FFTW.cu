#include "hip/hip_runtime.h"
//general parts
#include <stdio.h>
#include <vector>
#include <memory>
#include <string.h>
#include <chrono>
#include <thread>
#include <iostream>

//CUDA parts
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>

//Vulkan parts
#include <vulkan/vulkan.h>
#include <vkFFT.h>

//FFTW
#include "FFTW/fftw3.h"

#define GROUP 1

#ifdef NDEBUG
const bool enableValidationLayers = false;
#else
const bool enableValidationLayers = true;
#endif

VkInstance instance = {};
VkDebugReportCallbackEXT debugReportCallback = {};
VkPhysicalDevice physicalDevice = {};
VkPhysicalDeviceProperties physicalDeviceProperties = {};
VkPhysicalDeviceMemoryProperties physicalDeviceMemoryProperties = {};
VkDevice device = {};
VkDebugUtilsMessengerEXT debugMessenger = {};
uint32_t queueFamilyIndex = {};
std::vector<const char*> enabledLayers;
VkQueue queue = {};
VkCommandPool commandPool = {};
VkFence fence = {};

const std::vector<const char*> validationLayers = {
	"VK_LAYER_KHRONOS_validation"
};
static VKAPI_ATTR VkBool32 VKAPI_CALL debugReportCallbackFn(
	VkDebugReportFlagsEXT                       flags,
	VkDebugReportObjectTypeEXT                  objectType,
	uint64_t                                    object,
	size_t                                      location,
	int32_t                                     messageCode,
	const char* pLayerPrefix,
	const char* pMessage,
	void* pUserData) {

	printf("Debug Report: %s: %s\n", pLayerPrefix, pMessage);

	return VK_FALSE;
}

VkResult CreateDebugUtilsMessengerEXT(VkInstance instance, const VkDebugUtilsMessengerCreateInfoEXT* pCreateInfo, const VkAllocationCallbacks* pAllocator, VkDebugUtilsMessengerEXT* pDebugMessenger) {
	auto func = (PFN_vkCreateDebugUtilsMessengerEXT)vkGetInstanceProcAddr(instance, "vkCreateDebugUtilsMessengerEXT");
	if (func != nullptr) {
		return func(instance, pCreateInfo, pAllocator, pDebugMessenger);
	}
	else {
		return VK_ERROR_EXTENSION_NOT_PRESENT;
	}
}

void DestroyDebugUtilsMessengerEXT(VkInstance instance, VkDebugUtilsMessengerEXT debugMessenger, const VkAllocationCallbacks* pAllocator) {
	auto func = (PFN_vkDestroyDebugUtilsMessengerEXT)vkGetInstanceProcAddr(instance, "vkDestroyDebugUtilsMessengerEXT");
	if (func != nullptr) {
		func(instance, debugMessenger, pAllocator);
	}
}
static VKAPI_ATTR VkBool32 VKAPI_CALL debugCallback(VkDebugUtilsMessageSeverityFlagBitsEXT messageSeverity, VkDebugUtilsMessageTypeFlagsEXT messageType, const VkDebugUtilsMessengerCallbackDataEXT* pCallbackData, void* pUserData) {
	std::cerr << "validation layer: " << pCallbackData->pMessage << std::endl;

	return VK_FALSE;
}


void setupDebugMessenger() {
	if (!enableValidationLayers) return;

	VkDebugUtilsMessengerCreateInfoEXT createInfo = { VK_STRUCTURE_TYPE_DEBUG_UTILS_MESSENGER_CREATE_INFO_EXT };
	createInfo.messageSeverity = VK_DEBUG_UTILS_MESSAGE_SEVERITY_VERBOSE_BIT_EXT | VK_DEBUG_UTILS_MESSAGE_SEVERITY_WARNING_BIT_EXT | VK_DEBUG_UTILS_MESSAGE_SEVERITY_ERROR_BIT_EXT;
	createInfo.messageType = VK_DEBUG_UTILS_MESSAGE_TYPE_GENERAL_BIT_EXT | VK_DEBUG_UTILS_MESSAGE_TYPE_VALIDATION_BIT_EXT | VK_DEBUG_UTILS_MESSAGE_TYPE_PERFORMANCE_BIT_EXT;
	createInfo.pfnUserCallback = debugCallback;

	if (CreateDebugUtilsMessengerEXT(instance, &createInfo, nullptr, &debugMessenger) != VK_SUCCESS) {
		throw std::runtime_error("failed to set up debug messenger");
	}
}

std::vector<const char*> getRequiredExtensions() {
	std::vector<const char*> extensions;

	if (enableValidationLayers) {
		extensions.push_back(VK_EXT_DEBUG_UTILS_EXTENSION_NAME);
	}

	return extensions;
}

bool checkValidationLayerSupport() {
	uint32_t layerCount;
	vkEnumerateInstanceLayerProperties(&layerCount, nullptr);

	std::vector<VkLayerProperties> availableLayers(layerCount);
	vkEnumerateInstanceLayerProperties(&layerCount, availableLayers.data());

	for (const char* layerName : validationLayers) {
		bool layerFound = false;

		for (const auto& layerProperties : availableLayers) {
			if (strcmp(layerName, layerProperties.layerName) == 0) {
				layerFound = true;
				break;
			}
		}

		if (!layerFound) {
			return false;
		}
	}

	return true;
}

void createInstance() {
	if (enableValidationLayers && !checkValidationLayerSupport()) {
		throw std::runtime_error("validation layers creation failed");
	}

	VkApplicationInfo applicationInfo = { VK_STRUCTURE_TYPE_APPLICATION_INFO };
	applicationInfo.pApplicationName = "VkFFT";
	applicationInfo.applicationVersion = 1.0;
	applicationInfo.pEngineName = "VkFFT";
	applicationInfo.engineVersion = 1.0;
	applicationInfo.apiVersion = VK_API_VERSION_1_0;

	VkInstanceCreateInfo createInfo = { VK_STRUCTURE_TYPE_INSTANCE_CREATE_INFO };
	createInfo.flags = 0;
	createInfo.pApplicationInfo = &applicationInfo;

	auto extensions = getRequiredExtensions();
	createInfo.enabledExtensionCount = static_cast<uint32_t>(extensions.size());
	createInfo.ppEnabledExtensionNames = extensions.data();

	VkDebugUtilsMessengerCreateInfoEXT debugCreateInfo = { VK_STRUCTURE_TYPE_DEBUG_UTILS_MESSENGER_CREATE_INFO_EXT };
	if (enableValidationLayers) {
		createInfo.enabledLayerCount = static_cast<uint32_t>(validationLayers.size());
		createInfo.ppEnabledLayerNames = validationLayers.data();
		debugCreateInfo.messageSeverity = VK_DEBUG_UTILS_MESSAGE_SEVERITY_VERBOSE_BIT_EXT | VK_DEBUG_UTILS_MESSAGE_SEVERITY_WARNING_BIT_EXT | VK_DEBUG_UTILS_MESSAGE_SEVERITY_ERROR_BIT_EXT;
		debugCreateInfo.messageType = VK_DEBUG_UTILS_MESSAGE_TYPE_GENERAL_BIT_EXT | VK_DEBUG_UTILS_MESSAGE_TYPE_VALIDATION_BIT_EXT | VK_DEBUG_UTILS_MESSAGE_TYPE_PERFORMANCE_BIT_EXT;
		debugCreateInfo.pfnUserCallback = debugCallback;
		createInfo.pNext = (VkDebugUtilsMessengerCreateInfoEXT*)&debugCreateInfo;
	}
	else {
		createInfo.enabledLayerCount = 0;

		createInfo.pNext = nullptr;
	}

	if (vkCreateInstance(&createInfo, nullptr, &instance) != VK_SUCCESS) {
		throw std::runtime_error("instance creation failed");
	}


}

void findPhysicalDevice(uint32_t deviceID) {

	uint32_t deviceCount;
	vkEnumeratePhysicalDevices(instance, &deviceCount, NULL);
	if (deviceCount == 0) {
		throw std::runtime_error("device with vulkan support not found");
	}

	std::vector<VkPhysicalDevice> devices(deviceCount);
	vkEnumeratePhysicalDevices(instance, &deviceCount, devices.data());

	physicalDevice = devices[deviceID];

}

uint32_t getComputeQueueFamilyIndex() {
	uint32_t queueFamilyCount;

	vkGetPhysicalDeviceQueueFamilyProperties(physicalDevice, &queueFamilyCount, NULL);

	std::vector<VkQueueFamilyProperties> queueFamilies(queueFamilyCount);
	vkGetPhysicalDeviceQueueFamilyProperties(physicalDevice, &queueFamilyCount, queueFamilies.data());

	uint32_t i = 0;
	for (; i < queueFamilies.size(); ++i) {
		VkQueueFamilyProperties props = queueFamilies[i];

		if (props.queueCount > 0 && (props.queueFlags & VK_QUEUE_COMPUTE_BIT)) {
			break;
		}
	}

	if (i == queueFamilies.size()) {
		throw std::runtime_error("queue family creation failed");
	}

	return i;
}

void createDevice() {

	VkDeviceQueueCreateInfo queueCreateInfo = { VK_STRUCTURE_TYPE_DEVICE_QUEUE_CREATE_INFO };
	queueFamilyIndex = getComputeQueueFamilyIndex();
	queueCreateInfo.queueFamilyIndex = queueFamilyIndex;
	queueCreateInfo.queueCount = 1;
	float queuePriorities = 1.0;
	queueCreateInfo.pQueuePriorities = &queuePriorities;
	VkDeviceCreateInfo deviceCreateInfo = { VK_STRUCTURE_TYPE_DEVICE_CREATE_INFO };
	VkPhysicalDeviceFeatures deviceFeatures = {};
	deviceFeatures.shaderFloat64 = true;
	deviceCreateInfo.enabledLayerCount = enabledLayers.size();
	deviceCreateInfo.ppEnabledLayerNames = enabledLayers.data();
	deviceCreateInfo.pQueueCreateInfos = &queueCreateInfo;
	deviceCreateInfo.queueCreateInfoCount = 1;
	deviceCreateInfo.pEnabledFeatures = &deviceFeatures;
	vkCreateDevice(physicalDevice, &deviceCreateInfo, NULL, &device);
	vkGetDeviceQueue(device, queueFamilyIndex, 0, &queue);

}


uint32_t findMemoryType(uint32_t memoryTypeBits, VkMemoryPropertyFlags properties) {
	VkPhysicalDeviceMemoryProperties memoryProperties = {};

	vkGetPhysicalDeviceMemoryProperties(physicalDevice, &memoryProperties);

	for (uint32_t i = 0; i < memoryProperties.memoryTypeCount; ++i) {
		if ((memoryTypeBits & (1 << i)) &&
			((memoryProperties.memoryTypes[i].propertyFlags & properties) == properties))
			return i;
	}
	return -1;
}
void allocateFFTBuffer(VkBuffer* buffer, VkDeviceMemory* deviceMemory, VkBufferUsageFlags usageFlags, VkMemoryPropertyFlags propertyFlags, VkDeviceSize size) {
	uint32_t queueFamilyIndices;
	VkBufferCreateInfo bufferCreateInfo = { VK_STRUCTURE_TYPE_BUFFER_CREATE_INFO };
	bufferCreateInfo.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
	bufferCreateInfo.queueFamilyIndexCount = 1;
	bufferCreateInfo.pQueueFamilyIndices = &queueFamilyIndices;
	bufferCreateInfo.size = size;
	bufferCreateInfo.usage = usageFlags;
	vkCreateBuffer(device, &bufferCreateInfo, NULL, buffer);
	VkMemoryRequirements memoryRequirements = {};
	vkGetBufferMemoryRequirements(device, buffer[0], &memoryRequirements);
	VkMemoryAllocateInfo memoryAllocateInfo = { VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO };
	memoryAllocateInfo.allocationSize = memoryRequirements.size;
	memoryAllocateInfo.memoryTypeIndex = findMemoryType(memoryRequirements.memoryTypeBits, propertyFlags);
	vkAllocateMemory(device, &memoryAllocateInfo, NULL, deviceMemory);
	vkBindBufferMemory(device, buffer[0], deviceMemory[0], 0);
}
void transferDataFromCPU(float* arr, VkBuffer* buffer, VkDeviceSize bufferSize) {
	VkDeviceSize stagingBufferSize = bufferSize;
	VkBuffer stagingBuffer = {};
	VkDeviceMemory stagingBufferMemory = {};
	allocateFFTBuffer(&stagingBuffer, &stagingBufferMemory, VK_BUFFER_USAGE_TRANSFER_SRC_BIT, VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT, stagingBufferSize);

	void* data;
	vkMapMemory(device, stagingBufferMemory, 0, stagingBufferSize, 0, &data);
	memcpy(data, arr, stagingBufferSize);
	vkUnmapMemory(device, stagingBufferMemory);
	VkCommandBufferAllocateInfo commandBufferAllocateInfo = { VK_STRUCTURE_TYPE_COMMAND_BUFFER_ALLOCATE_INFO };
	commandBufferAllocateInfo.commandPool = commandPool;
	commandBufferAllocateInfo.level = VK_COMMAND_BUFFER_LEVEL_PRIMARY;
	commandBufferAllocateInfo.commandBufferCount = 1;
	VkCommandBuffer commandBuffer = {};
	vkAllocateCommandBuffers(device, &commandBufferAllocateInfo, &commandBuffer);
	VkCommandBufferBeginInfo commandBufferBeginInfo = { VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO };
	commandBufferBeginInfo.flags = VK_COMMAND_BUFFER_USAGE_ONE_TIME_SUBMIT_BIT;
	vkBeginCommandBuffer(commandBuffer, &commandBufferBeginInfo);
	VkBufferCopy copyRegion = {};
	copyRegion.srcOffset = 0;
	copyRegion.dstOffset = 0;
	copyRegion.size = stagingBufferSize;
	vkCmdCopyBuffer(commandBuffer, stagingBuffer, buffer[0], 1, &copyRegion);
	vkEndCommandBuffer(commandBuffer);
	VkSubmitInfo submitInfo = { VK_STRUCTURE_TYPE_SUBMIT_INFO };
	submitInfo.commandBufferCount = 1;
	submitInfo.pCommandBuffers = &commandBuffer;
	vkQueueSubmit(queue, 1, &submitInfo, fence);
	vkWaitForFences(device, 1, &fence, VK_TRUE, 100000000000);
	vkResetFences(device, 1, &fence);
	vkFreeCommandBuffers(device, commandPool, 1, &commandBuffer);
	vkDestroyBuffer(device, stagingBuffer, NULL);
	vkFreeMemory(device, stagingBufferMemory, NULL);
}
void transferDataToCPU(float* arr, VkBuffer* buffer, VkDeviceSize bufferSize) {
	VkDeviceSize stagingBufferSize = bufferSize;
	VkBuffer stagingBuffer = {};
	VkDeviceMemory stagingBufferMemory = {};
	allocateFFTBuffer(&stagingBuffer, &stagingBufferMemory, VK_BUFFER_USAGE_TRANSFER_DST_BIT, VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT, stagingBufferSize);


	VkCommandBufferAllocateInfo commandBufferAllocateInfo = { VK_STRUCTURE_TYPE_COMMAND_BUFFER_ALLOCATE_INFO };
	commandBufferAllocateInfo.commandPool = commandPool;
	commandBufferAllocateInfo.level = VK_COMMAND_BUFFER_LEVEL_PRIMARY;
	commandBufferAllocateInfo.commandBufferCount = 1;
	VkCommandBuffer commandBuffer = {};
	vkAllocateCommandBuffers(device, &commandBufferAllocateInfo, &commandBuffer);
	VkCommandBufferBeginInfo commandBufferBeginInfo = { VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO };
	commandBufferBeginInfo.flags = VK_COMMAND_BUFFER_USAGE_ONE_TIME_SUBMIT_BIT;
	vkBeginCommandBuffer(commandBuffer, &commandBufferBeginInfo);
	VkBufferCopy copyRegion = {};
	copyRegion.srcOffset = 0;
	copyRegion.dstOffset = 0;
	copyRegion.size = stagingBufferSize;
	vkCmdCopyBuffer(commandBuffer, buffer[0], stagingBuffer, 1, &copyRegion);
	vkEndCommandBuffer(commandBuffer);
	VkSubmitInfo submitInfo = { VK_STRUCTURE_TYPE_SUBMIT_INFO };
	submitInfo.commandBufferCount = 1;
	submitInfo.pCommandBuffers = &commandBuffer;
	vkQueueSubmit(queue, 1, &submitInfo, fence);
	vkWaitForFences(device, 1, &fence, VK_TRUE, 100000000000);
	vkResetFences(device, 1, &fence);
	vkFreeCommandBuffers(device, commandPool, 1, &commandBuffer);
	void* data;
	vkMapMemory(device, stagingBufferMemory, 0, stagingBufferSize, 0, &data);
	memcpy(arr, data, stagingBufferSize);
	vkUnmapMemory(device, stagingBufferMemory);
	vkDestroyBuffer(device, stagingBuffer, NULL);
	vkFreeMemory(device, stagingBufferMemory, NULL);
}

float performVulkanFFT(VkFFTApplication* app_forward, uint32_t batch) {
	VkCommandBufferAllocateInfo commandBufferAllocateInfo = { VK_STRUCTURE_TYPE_COMMAND_BUFFER_ALLOCATE_INFO };
	commandBufferAllocateInfo.commandPool = commandPool;
	commandBufferAllocateInfo.level = VK_COMMAND_BUFFER_LEVEL_PRIMARY;
	commandBufferAllocateInfo.commandBufferCount = 1;
	VkCommandBuffer commandBuffer = {};
	vkAllocateCommandBuffers(device, &commandBufferAllocateInfo, &commandBuffer);
	VkCommandBufferBeginInfo commandBufferBeginInfo = { VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO };
	commandBufferBeginInfo.flags = VK_COMMAND_BUFFER_USAGE_ONE_TIME_SUBMIT_BIT;
	vkBeginCommandBuffer(commandBuffer, &commandBufferBeginInfo);
	for (uint32_t i = 0; i < batch; i++) {
		app_forward->VkFFTAppend(commandBuffer);
	}
	vkEndCommandBuffer(commandBuffer);
	VkSubmitInfo submitInfo = { VK_STRUCTURE_TYPE_SUBMIT_INFO };
	submitInfo.commandBufferCount = 1;
	submitInfo.pCommandBuffers = &commandBuffer;
	auto timeSubmit = std::chrono::steady_clock::now();
	vkQueueSubmit(queue, 1, &submitInfo, fence);
	vkWaitForFences(device, 1, &fence, VK_TRUE, 100000000000);
	auto timeEnd = std::chrono::steady_clock::now();
	float totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001;
	vkResetFences(device, 1, &fence);
	vkFreeCommandBuffers(device, commandPool, 1, &commandBuffer);
	return totTime / batch;
}

int main()
{
    createInstance();
    setupDebugMessenger();
    findPhysicalDevice(0);
    createDevice();

    VkFenceCreateInfo fenceCreateInfo = { VK_STRUCTURE_TYPE_FENCE_CREATE_INFO };
    fenceCreateInfo.flags = 0;
    vkCreateFence(device, &fenceCreateInfo, NULL, &fence);
    VkCommandPoolCreateInfo commandPoolCreateInfo = { VK_STRUCTURE_TYPE_COMMAND_POOL_CREATE_INFO };
    commandPoolCreateInfo.flags = VK_COMMAND_POOL_CREATE_RESET_COMMAND_BUFFER_BIT;
    commandPoolCreateInfo.queueFamilyIndex = queueFamilyIndex;
    vkCreateCommandPool(device, &commandPoolCreateInfo, NULL, &commandPool);
    vkGetPhysicalDeviceProperties(physicalDevice, &physicalDeviceProperties);
    vkGetPhysicalDeviceMemoryProperties(physicalDevice, &physicalDeviceMemoryProperties);

    const int num_benchmark_samples = 47;
    const int num_runs = 1;

	int benchmark_dimensions[num_benchmark_samples][4] = { {32, 1, 1, 1}, { 256, 1, 1, 1 }, { 1024, 1, 1, 1 }, { 4096, 1, 1, 1 }, { 8192, 1, 1, 1 }, { 16384, 1, 1, 1 },
		{32, 32, 1, 2},  {64, 64, 1, 2}, {256, 32, 1, 2}, {32, 256, 1, 2}, {256, 256, 1, 2}, {1024, 256, 1, 2},{256, 1024, 1, 2}, {512, 512, 1, 2}, {1024, 1024, 1, 2} , {4096, 1024, 1, 2},
		{32, 32, 32, 3}, {256, 32, 8, 3},{8, 32, 256, 3}, {32, 8, 256, 3}, {512, 256, 32, 3}, {1024, 1024, 8, 3}, {4096, 256, 8, 3},
		{4096, 4096, 1, 2}, {4096, 4096, 8, 3}, {1024, 4096, 1, 2}, {256, 8, 4096, 3},
		{(uint32_t)pow(2,15), 1, 1, 1}, {(uint32_t)pow(2,16), 1, 1, 1},  {(uint32_t)pow(2,17), 1, 1, 1},  {(uint32_t)pow(2,18), 1, 1, 1},   {(uint32_t)pow(2,20), 1, 1, 1},   {(uint32_t)pow(2,22), 1, 1, 1}, 
		{(uint32_t)pow(2,15), 64, 1, 2}, {(uint32_t)pow(2,16), 64, 1, 2}, {(uint32_t)pow(2,17), 64, 1, 2}, {(uint32_t)pow(2,18), 64, 1, 2},  {(uint32_t)pow(2,20), 64, 1, 2},  {(uint32_t)pow(2,22), 64, 1, 2}, 
		{64, (uint32_t)pow(2,13), 1, 2}, {64, (uint32_t)pow(2,14), 1, 2}, {64, (uint32_t)pow(2,15), 1, 2}, 
		{8,8 , (uint32_t)pow(2,13), 3}, {8,8, (uint32_t)pow(2,14), 3}, {8,8, (uint32_t)pow(2,15), 3},
		{(uint32_t)pow(2,13), (uint32_t)pow(2,13), 1, 2},{(uint32_t)pow(2,14), (uint32_t)pow(2,14), 1, 2}
	};
   
    double benchmark_result = 0;//averaged result = sum(system_size/iteration_time)/num_benchmark_samples

    for (int n = 0; n < num_benchmark_samples; n++) {
        for (int r = 0; r < num_runs; r++) {
            hipfftHandle planC2C;
            hipfftComplex* dataC;
            hipfftComplex* inputC;
			fftw_complex* inputC_double;
            int dims[3] = { benchmark_dimensions[n][0] , benchmark_dimensions[n][1] ,benchmark_dimensions[n][2] };

            inputC = (hipfftComplex*)(malloc(sizeof(hipfftComplex) * dims[0] * dims[1] * dims[2]));
			inputC_double = (fftw_complex*)(malloc(sizeof(fftw_complex) * dims[0] * dims[1] * dims[2]));
			for (int l = 0; l < dims[2]; l++) {
				for (int j = 0; j < dims[1]; j++) {
					for (int i = 0; i < dims[0]; i++) {
						inputC[i + j * dims[0] + l * dims[0] * dims[1]].x = 2 * ((float)rand()) / RAND_MAX - 1.0;
						inputC[i + j * dims[0] + l * dims[0] * dims[1]].y = 2 * ((float)rand()) / RAND_MAX - 1.0;
						inputC_double[i + j * dims[0] + l * dims[0] * dims[1]][0] = (double)inputC[i + j * dims[0] + l * dims[0] * dims[1]].x;
						inputC_double[i + j * dims[0] + l * dims[0] * dims[1]][1] = (double)inputC[i + j * dims[0] + l * dims[0] * dims[1]].y;
					}
				}
			}
			
			fftw_plan p;
			
			fftw_complex* output_FFTW = (fftw_complex*)(malloc(sizeof(fftw_complex) * dims[0] * dims[1] * dims[2]));
			
			switch (benchmark_dimensions[n][3]) {
			case 1:
				p = fftw_plan_dft_1d(benchmark_dimensions[n][0], inputC_double, output_FFTW, 1, FFTW_ESTIMATE);
				break;
			case 2:
				p = fftw_plan_dft_2d(benchmark_dimensions[n][1], benchmark_dimensions[n][0], inputC_double, output_FFTW, 1, FFTW_ESTIMATE);
				break;
			case 3:
				p = fftw_plan_dft_3d(benchmark_dimensions[n][2], benchmark_dimensions[n][1], benchmark_dimensions[n][0], inputC_double, output_FFTW, 1, FFTW_ESTIMATE);
				break;
			}

			fftw_execute(p); 

            hipMalloc((void**)&dataC, sizeof(hipfftComplex) * dims[0] * dims[1]*dims[2]);
            
            hipMemcpy(dataC, inputC, sizeof(hipfftComplex) * dims[0] * dims[1] * dims[2], hipMemcpyHostToDevice);
            if (hipGetLastError() != hipSuccess) {
                fprintf(stderr, "Cuda error: Failed to allocate\n");
                return;
            }
			switch (benchmark_dimensions[n][3]) {
			case 1:
				hipfftPlan1d(&planC2C, dims[0], HIPFFT_C2C, 1);
				break;
			case 2:
				hipfftPlan2d(&planC2C, dims[1], dims[0], HIPFFT_C2C);
				break;
			case 3:
				hipfftPlan3d(&planC2C, dims[2], dims[1], dims[0], HIPFFT_C2C);
				break;
			}
			
            float totTime = 0;
            int batch = 1;
            auto timeSubmit = std::chrono::steady_clock::now();
            hipDeviceSynchronize();
            for (int i = 0; i < batch; i++) {

                hipfftExecC2C(planC2C, dataC, dataC, 1);

            }
            hipDeviceSynchronize();
            auto timeEnd = std::chrono::steady_clock::now();
            totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001) / batch;

            hipfftComplex* output_cuFFT = (hipfftComplex*)(malloc(sizeof(hipfftComplex) * dims[0] * dims[1]*dims[2]));
            hipMemcpy(output_cuFFT, dataC, sizeof(hipfftComplex) * dims[0] * dims[1] * dims[2], hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

			//VkFFT part

			VkFFTConfiguration forward_configuration;
			VkFFTApplication app_forward;
			//VkFFTApplication app_inverse;
			forward_configuration.coalescedMemory = 32;//in bits, for Nvidia compute capability >=6.0 is equal to 32, <6.0 is equal 128. For Intel use 64. Gonna work regardles, but if specified by user correctly, the performance will be higher. 

			forward_configuration.FFTdim = benchmark_dimensions[n][3]; //FFT dimension, 1D, 2D or 3D (default 1).
			forward_configuration.size[0] = benchmark_dimensions[n][0]; //Multidimensional FFT dimensions sizes (default 1). For best performance (and stability), order dimensions in descendant size order as: x>y>z.   
			forward_configuration.size[1] = benchmark_dimensions[n][1];
			forward_configuration.size[2] = benchmark_dimensions[n][2];
			//registerBoost should be disabled on machines with <256KB register file
			forward_configuration.registerBoost = 4;
			forward_configuration.performZeropadding[0] = false; //Perform padding with zeros on GPU. Still need to properly align input data (no need to fill padding area with meaningful data) but this will increase performance due to the lower amount of the memory reads/writes and omitting sequences only consisting of zeros.
			forward_configuration.performZeropadding[1] = false;
			forward_configuration.performZeropadding[2] = false;
			forward_configuration.performConvolution = false; //Perform convolution with precomputed kernel. 
			forward_configuration.performR2C = false; //Perform C2C transform. Can be combined with all other options. 
			forward_configuration.coordinateFeatures = 1; //Specify dimensionality of the input feature vector (default 1). Each component is stored not as a vector, but as a separate system and padded on it's own according to other options (i.e. for x*y system of 3-vector, first x*y elements correspond to the first dimension, then goes x*y for the second, etc). 
			forward_configuration.inverse = false; //Direction of FFT. false - forward, true - inverse.
			//After this, configuration file contains pointers to Vulkan objects needed to work with the GPU: VkDevice* device - created device, [VkDeviceSize *bufferSize, VkBuffer *buffer, VkDeviceMemory* bufferDeviceMemory] - allocated GPU memory FFT is performed on. [VkDeviceSize *kernelSize, VkBuffer *kernel, VkDeviceMemory* kernelDeviceMemory] - allocated GPU memory, where kernel for convolution is stored.
			forward_configuration.device = &device;
			//Custom path to the floder with shaders, default is "shaders");
			sprintf(forward_configuration.shaderPath, "shaders\\");

			//Allocate buffer for the input data.
			VkDeviceSize bufferSize = forward_configuration.coordinateFeatures * sizeof(float) * 2 * forward_configuration.size[0] * forward_configuration.size[1] * forward_configuration.size[2];;
			VkBuffer buffer = {};
			VkDeviceMemory bufferDeviceMemory = {};

			allocateFFTBuffer(&buffer, &bufferDeviceMemory, VK_BUFFER_USAGE_STORAGE_BUFFER_BIT | VK_BUFFER_USAGE_TRANSFER_SRC_BIT | VK_BUFFER_USAGE_TRANSFER_DST_BIT, VK_MEMORY_HEAP_DEVICE_LOCAL_BIT, bufferSize);
			forward_configuration.buffer = &buffer;
			forward_configuration.isInputFormatted = false; //set to true if input is a different buffer, so it can have zeropadding/R2C added  
			forward_configuration.inputBuffer = &buffer; //you can specify first buffer to read data from to be different from the buffer FFT is performed on. FFT is still in-place on the second buffer, this is here just for convenience.
			forward_configuration.isOutputFormatted = false;//set to true if output is a different buffer, so it can have zeropadding/C2R automatically removed
			forward_configuration.outputBuffer = &buffer;
			forward_configuration.bufferSize = &bufferSize;
			forward_configuration.inputBufferSize = &bufferSize;
			forward_configuration.outputBufferSize = &bufferSize;

			//Sample buffer transfer tool. Uses staging buffer of the same size as destination buffer, which can be reduced if transfer is done sequentially in small buffers.
			transferDataFromCPU((float*) inputC, &buffer, bufferSize);
			//Initialize applications. This function loads shaders, creates pipeline and configures FFT based on configuration file. No buffer allocations inside VkFFT library.  
			app_forward.initializeVulkanFFT(forward_configuration);
			//forward_configuration.inverse = true;
			//app_inverse.initializeVulkanFFT(forward_configuration);
			//Submit FFT+iFFT.
			//batch = 1;
			totTime = performVulkanFFT(&app_forward, batch);
			//totTime = performVulkanFFT(&app_inverse, batch);
			hipfftComplex* output_VkFFT = (hipfftComplex*)malloc(bufferSize);

			//Transfer data from GPU using staging buffer.
			transferDataToCPU((float*)output_VkFFT, &buffer, bufferSize);

			float avg_difference[2] = { 0,0 };
			float max_difference[2] = { 0,0 };
			float avg_eps[2] = { 0,0 };
			float max_eps[2] = { 0,0 };
			for (int l = 0; l < dims[2]; l++) {
				for (int j = 0; j < dims[1]; j++) {
					for (int i = 0; i < dims[0]; i++) {
						int loc_i = i;
						int loc_j = j;
						int loc_l = l;
						//VkFFT doesn't reshuffle after 4 step FFT
						if (app_forward.localFFTPlan.numAxisUploads[0] > 1)
							loc_i = i / app_forward.localFFTPlan.axes[0][1].specializationConstants.fftDim + app_forward.localFFTPlan.axes[0][0].specializationConstants.fftDim*(i % app_forward.localFFTPlan.axes[0][1].specializationConstants.fftDim);
						if (app_forward.localFFTPlan.numAxisUploads[1] > 1)
							loc_j = j / app_forward.localFFTPlan.axes[1][1].specializationConstants.fftDim + app_forward.localFFTPlan.axes[1][0].specializationConstants.fftDim * (j % app_forward.localFFTPlan.axes[1][1].specializationConstants.fftDim);
						if (app_forward.localFFTPlan.numAxisUploads[2] > 1)
							loc_l = l / app_forward.localFFTPlan.axes[2][1].specializationConstants.fftDim + app_forward.localFFTPlan.axes[2][0].specializationConstants.fftDim * (l % app_forward.localFFTPlan.axes[2][1].specializationConstants.fftDim);

						//printf("%f %f - %f %f - %f %f\n", output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][0], output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][1], output_cuFFT[i + j * dims[0] + l * dims[0] * dims[1]].x, output_cuFFT[i + j * dims[0] + l * dims[0] * dims[1]].y, output_VkFFT[(loc_i + loc_j * dims[0]+ loc_l * dims[0] * dims[1])].x, output_VkFFT[(loc_i + loc_j * dims[0]+ loc_l * dims[0] * dims[1])].y);

						float current_diff_x_cuFFT = (output_cuFFT[i + j * dims[0] + l * dims[0] * dims[1]].x - output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][0]);
						float current_diff_y_cuFFT = (output_cuFFT[i + j * dims[0] + l * dims[0] * dims[1]].y - output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][1]);
						float current_diff_x_VkFFT = (output_VkFFT[loc_i + loc_j * dims[0] + loc_l * dims[0] * dims[1]].x - output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][0]);
						float current_diff_y_VkFFT = (output_VkFFT[loc_i + loc_j * dims[0] + loc_l * dims[0] * dims[1]].y - output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][1]);

						float current_diff_norm_cuFFT = sqrt(current_diff_x_cuFFT * current_diff_x_cuFFT + current_diff_y_cuFFT * current_diff_y_cuFFT);
						float current_diff_norm_VkFFT = sqrt(current_diff_x_VkFFT * current_diff_x_VkFFT + current_diff_y_VkFFT * current_diff_y_VkFFT);
						float current_data_norm = sqrt(output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][0] * output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][0] + output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][1] * output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][1]);
						if (current_diff_norm_cuFFT > max_difference[0]) max_difference[0] = current_diff_norm_cuFFT;
						avg_difference[0] += current_diff_norm_cuFFT;

						if ((current_diff_norm_cuFFT / current_data_norm > max_eps[0]) && (current_data_norm > 1e-10)) {
							//printf("%f %f - %f %f\n", output_cuFFT[i + j * dims[0]].x, output_cuFFT[i + j * dims[0]].y, output_VkFFT[i + j * dims[0]].x, output_VkFFT[i + j * dims[0]].y);

							max_eps[0] = current_diff_norm_cuFFT / current_data_norm;
						}
						avg_eps[0] += (current_data_norm > 1e-10) ? current_diff_norm_cuFFT / current_data_norm : 0;

						if (current_diff_norm_VkFFT > max_difference[1]) max_difference[1] = current_diff_norm_VkFFT;
						avg_difference[1] += current_diff_norm_VkFFT;

						if ((current_diff_norm_VkFFT / current_data_norm > max_eps[1]) && (current_data_norm > 1e-10)) {
							//printf("%f %f - %f %f\n", output_cuFFT[i + j * dims[0]].x, output_cuFFT[i + j * dims[0]].y, output_VkFFT[i + j * dims[0]].x, output_VkFFT[i + j * dims[0]].y);

							max_eps[1] = current_diff_norm_VkFFT / current_data_norm;
						}
						avg_eps[1] += (current_data_norm > 1e-10) ? current_diff_norm_VkFFT / current_data_norm : 0;
					}
					//printf("\n");
				}
			}
			avg_difference[0] /= (dims[0] * dims[1]*dims[2]);
			avg_eps[0] /= (dims[0] * dims[1]*dims[2]);
			avg_difference[1] /= (dims[0] * dims[1] * dims[2]);
			avg_eps[1] /= (dims[0] * dims[1] * dims[2]);
			printf("cuFFT System: %dx%dx%d avg_difference: %f max_difference: %f avg_eps: %f max_eps: %f\n", dims[0], dims[1], dims[2], avg_difference[0], max_difference[0], avg_eps[0], max_eps[0]);
			printf("VkFFT System: %dx%dx%d avg_difference: %f max_difference: %f avg_eps: %f max_eps: %f\n", dims[0], dims[1], dims[2], avg_difference[1], max_difference[1], avg_eps[1], max_eps[1]);
			free(output_cuFFT);
			free(output_VkFFT);
			vkDestroyBuffer(device, buffer, NULL);
			vkFreeMemory(device, bufferDeviceMemory, NULL);
			app_forward.deleteVulkanFFT();
            hipfftDestroy(planC2C);
            hipFree(dataC);
            free(inputC);
			fftw_destroy_plan(p);
			free(inputC_double);
			free(output_FFTW);
        }
    }
	
}
