//general parts
#include <stdio.h>
#include <vector>
#include <memory>
#include <string.h>
#include <chrono>
#include <thread>
#include <iostream>

//CUDA parts
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define GROUP 1


void launch_benchmark_cuFFT_single_3d(bool file_output, FILE* output)
{
	if (file_output)
		fprintf(output, "3 - cuFFT FFT + iFFT C2C multidimensional benchmark in single precision\n");
	printf("3 - cuFFT FFT + iFFT C2C multidimensional benchmark in single precision\n");
	const int num_benchmark_samples = 16;
	const int num_runs = 3;

	uint32_t benchmark_dimensions[num_benchmark_samples][4] = { {1024, 1024, 1, 2},
		{(uint32_t)pow(2,5), (uint32_t)pow(2,5), 1, 2},{(uint32_t)pow(2,6), (uint32_t)pow(2,6), 1, 2}, {(uint32_t)pow(2,7), (uint32_t)pow(2,7), 1, 2},{(uint32_t)pow(2,8), (uint32_t)pow(2,8), 1, 2},
		{(uint32_t)pow(2,9), (uint32_t)pow(2,9), 1, 2},{(uint32_t)pow(2,10), (uint32_t)pow(2,10), 1, 2},{(uint32_t)pow(2,11), (uint32_t)pow(2,11), 1, 2},{(uint32_t)pow(2,12), (uint32_t)pow(2,12), 1, 2},
		{(uint32_t)pow(2,13), (uint32_t)pow(2,13), 1, 2},{(uint32_t)pow(2,14), (uint32_t)pow(2,13), 1, 2},
		{(uint32_t)pow(2,5), (uint32_t)pow(2,5), (uint32_t)pow(2,5), 3},{(uint32_t)pow(2,6), (uint32_t)pow(2,6), (uint32_t)pow(2,6), 3},{(uint32_t)pow(2,7), (uint32_t)pow(2,7), (uint32_t)pow(2,7), 3},
		{(uint32_t)pow(2,8), (uint32_t)pow(2,8), (uint32_t)pow(2,8), 3},{(uint32_t)pow(2,9), (uint32_t)pow(2,9), (uint32_t)pow(2,9), 3},
	};

	double benchmark_result[2] = { 0,0 };//averaged result = sum(system_size/iteration_time)/num_benchmark_samples
	hipfftComplex* inputC = (hipfftComplex*)malloc((uint64_t)sizeof(hipfftComplex)*pow(2, 27));
	for (uint64_t i = 0; i < pow(2, 27); i++) {
		inputC[i].x = 2 * ((float)rand()) / RAND_MAX - 1.0;
		inputC[i].y = 2 * ((float)rand()) / RAND_MAX - 1.0;
	}
	for (int n = 0; n < num_benchmark_samples; n++) {
		double run_time[num_runs][2];
		for (int r = 0; r < num_runs; r++) {
			hipfftHandle planC2C;
			hipfftComplex* dataC;

			uint32_t dims[3] = { benchmark_dimensions[n][0] , benchmark_dimensions[n][1] ,benchmark_dimensions[n][2] };

			hipMalloc((void**)&dataC, sizeof(hipfftComplex) * dims[0] * dims[1] * dims[2]);

			hipMemcpy(dataC, inputC, sizeof(hipfftComplex) * dims[0] * dims[1] * dims[2], hipMemcpyHostToDevice);
			if (hipGetLastError() != hipSuccess) {
				fprintf(stderr, "Cuda error: Failed to allocate\n");
				return;
			}
			switch (benchmark_dimensions[n][3]) {
			case 1:
				hipfftPlan1d(&planC2C, dims[0], HIPFFT_C2C, 1);
				break;
			case 2:
				hipfftPlan2d(&planC2C, dims[1], dims[0], HIPFFT_C2C);
				break;
			case 3:
				hipfftPlan3d(&planC2C, dims[2], dims[1], dims[0], HIPFFT_C2C);
				break;
			}

			float totTime = 0;
			uint32_t cuBufferSize = sizeof(float) * 2 * dims[0] * dims[1] * dims[2];
			uint32_t batch = ((4096 * 1024.0 * 1024.0) / cuBufferSize > 1000) ? 1000 : (4096 * 1024.0 * 1024.0) / cuBufferSize;
			if (batch == 0) batch = 1;
			auto timeSubmit = std::chrono::steady_clock::now();
			for (int i = 0; i < batch; i++) {

				hipfftExecC2C(planC2C, dataC, dataC, 1);
				hipfftExecC2C(planC2C, dataC, dataC, -1);
			}
			hipDeviceSynchronize();
			auto timeEnd = std::chrono::steady_clock::now();
			totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001) / batch;
			run_time[r][0] = totTime;
			if (n > 0) {
				if (r == num_runs - 1) {
					double std_error = 0;
					double avg_time = 0;
					for (uint32_t t = 0; t < num_runs; t++) {
						avg_time += run_time[t][0];
					}
					avg_time /= num_runs;
					for (uint32_t t = 0; t < num_runs; t++) {
						std_error += (run_time[t][0] - avg_time) * (run_time[t][0] - avg_time);
					}
					std_error = sqrt(std_error / num_runs);
					if (file_output)
						fprintf(output, "cuFFT System: %dx%dx%d Buffer: %d MB avg_time_per_step: %0.3f ms std_error: %0.3f batch: %d benchmark: %d\n", benchmark_dimensions[n][0], benchmark_dimensions[n][1], benchmark_dimensions[n][2], cuBufferSize / 1024 / 1024, avg_time, std_error, batch, (int)(((double)cuBufferSize / 1024) / avg_time));

					printf("cuFFT System: %dx%dx%d Buffer: %d MB avg_time_per_step: %0.3f ms std_error: %0.3f batch: %d benchmark: %d\n", benchmark_dimensions[n][0], benchmark_dimensions[n][1], benchmark_dimensions[n][2], cuBufferSize / 1024 / 1024, avg_time, std_error, batch, (int)(((double)cuBufferSize / 1024) / avg_time));
					benchmark_result[0] += ((double)cuBufferSize / 1024) / avg_time;
				}

			}
			hipfftDestroy(planC2C);
			hipFree(dataC);
			hipDeviceSynchronize();
			//cufftComplex* output_cuFFT = (cufftComplex*)(malloc(sizeof(cufftComplex) * dims[0] * dims[1] * dims[2]));
			//cudaMemcpy(output_cuFFT, dataC, sizeof(cufftComplex) * dims[0] * dims[1] * dims[2], cudaMemcpyDeviceToHost);
			//cudaDeviceSynchronize();
			

		}
	}
	free(inputC);
	benchmark_result[0] /= (num_benchmark_samples - 1);
	if (file_output)
		fprintf(output, "Benchmark score cuFFT: %d\n", (int)(benchmark_result[0]));
	printf("Benchmark score cuFFT: %d\n", (int)(benchmark_result[0]));

}
