//general parts
#include <stdio.h>
#include <vector>
#include <memory>
#include <string.h>
#include <chrono>
#include <thread>
#include <iostream>
#ifndef __STDC_FORMAT_MACROS
#define __STDC_FORMAT_MACROS
#endif
#include <inttypes.h>

//CUDA parts
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define GROUP 1


void sample_0_benchmark_cuFFT_single(bool file_output, FILE* output)
{
	
	const int num_runs = 3;
	if (file_output)
		fprintf(output, "0 - cuFFT FFT + iFFT C2C benchmark 1D batched in single precision\n");
	printf("0 - cuFFT FFT + iFFT C2C benchmark 1D batched in single precision\n");
	double benchmark_result[2] = { 0,0 };//averaged result = sum(system_size/iteration_time)/num_benchmark_samples
	hipfftComplex* inputC = (hipfftComplex*)malloc((uint64_t)sizeof(hipfftComplex)*pow(2, 27));
	for (uint64_t i = 0; i < pow(2, 27); i++) {
		inputC[i].x = 2 * ((float)rand()) / RAND_MAX - 1.0;
		inputC[i].y = 2 * ((float)rand()) / RAND_MAX - 1.0;
	}
	for (int n = 0; n < 26; n++) {
		double run_time[num_runs][2];
		for (int r = 0; r < num_runs; r++) {
			hipfftHandle planC2C;
			hipfftComplex* dataC;

			uint64_t dims[3];
			dims[0] = 4 * pow(2, n); //Multidimensional FFT dimensions sizes (default 1). For best performance (and stability), order dimensions in descendant size order as: x>y>z.   
			if (n == 0) dims[0] = 4096; 
			dims[1] = 64* 32 * pow(2, 16)/dims[0];
			//dims[1] = (dims[1] > 32768) ? 32768 : dims[1];
			if (dims[1] == 0) dims[1] = 1;
			dims[2] = 1;
			
			hipMalloc((void**)&dataC, sizeof(hipfftComplex) * dims[0] * dims[1] * dims[2]);

			hipMemcpy(dataC, inputC, sizeof(hipfftComplex) * dims[0] * dims[1] * dims[2], hipMemcpyHostToDevice);
			if (hipGetLastError() != hipSuccess) {
				fprintf(stderr, "Cuda error: Failed to allocate\n");
				return;
			}
			uint64_t sizeCUDA;
			switch (1) {
			case 1:
				hipfftPlan1d(&planC2C, dims[0], HIPFFT_C2C, dims[1]);
				hipfftGetSize1d(planC2C, dims[0], HIPFFT_C2C, dims[1], (size_t*)&sizeCUDA);
				break;
			case 2:
				hipfftPlan2d(&planC2C, dims[1], dims[0], HIPFFT_C2C);
				break;
			case 3:
				hipfftPlan3d(&planC2C, dims[2], dims[1], dims[0], HIPFFT_C2C);
				break;
			}

			float totTime = 0;
			uint64_t cuBufferSize = sizeof(float) * 2 * dims[0] * dims[1] * dims[2];
			uint64_t num_iter = ((3*4096 * 1024.0 * 1024.0) / cuBufferSize > 1000) ? 1000 : (3*4096 * 1024.0 * 1024.0) / cuBufferSize;
			if (num_iter == 0) num_iter = 1;
			
			std::chrono::steady_clock::time_point timeSubmit = std::chrono::steady_clock::now();
			for (int i = 0; i < num_iter; i++) {

				hipfftExecC2C(planC2C, dataC, dataC, -1);
				hipfftExecC2C(planC2C, dataC, dataC, 1);
			}
			hipDeviceSynchronize();
			std::chrono::steady_clock::time_point timeEnd = std::chrono::steady_clock::now();
			totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001) / num_iter;
			run_time[r][0] = totTime;
			if (n > 0) {
				if (r == num_runs - 1) {
					double std_error = 0;
					double avg_time = 0;
					for (uint64_t t = 0; t < num_runs; t++) {
						avg_time += run_time[t][0];
					}
					avg_time /= num_runs;
					for (uint64_t t = 0; t < num_runs; t++) {
						std_error += (run_time[t][0] - avg_time) * (run_time[t][0] - avg_time);
					}
					std_error = sqrt(std_error / num_runs);
					if (file_output)
						fprintf(output, "cuFFT System: %" PRIu64 " %" PRIu64 "x%" PRIu64 " Buffer: %" PRIu64 " MB avg_time_per_step: %0.3f ms std_error: %0.3f num_iter: %" PRIu64 " benchmark: %" PRIu64 "\n", (uint64_t)log2(dims[0]), dims[0], dims[1], cuBufferSize / 1024 / 1024, avg_time, std_error, num_iter, (uint64_t)(((double)cuBufferSize / 1024) / avg_time));

					printf("cuFFT System: %" PRIu64 " %" PRIu64 "x%" PRIu64 " Buffer: %" PRIu64 " MB avg_time_per_step: %0.3f ms std_error: %0.3f num_iter: %" PRIu64 " benchmark: %" PRIu64 "\n", (uint64_t)log2(dims[0]), dims[0], dims[1], cuBufferSize / 1024 / 1024, avg_time, std_error, num_iter, (uint64_t)(((double)cuBufferSize / 1024) / avg_time));
					benchmark_result[0] += ((double)cuBufferSize / 1024) / avg_time;
				}

			}
			hipfftDestroy(planC2C);
			hipFree(dataC);
			hipDeviceSynchronize();
			//cufftComplex* output_cuFFT = (cufftComplex*)(malloc(sizeof(cufftComplex) * dims[0] * dims[1] * dims[2]));
			//cudaMemcpy(output_cuFFT, dataC, sizeof(cufftComplex) * dims[0] * dims[1] * dims[2], cudaMemcpyDeviceToHost);
			//cudaDeviceSynchronize();
			

		}
	}
	free(inputC);
	benchmark_result[0] /= (26 - 1);
	if (file_output)
		fprintf(output, "Benchmark score cuFFT: %" PRIu64 "\n", (uint64_t)(benchmark_result[0]));
	printf("Benchmark score cuFFT: %" PRIu64 "\n", (uint64_t)(benchmark_result[0]));

}
