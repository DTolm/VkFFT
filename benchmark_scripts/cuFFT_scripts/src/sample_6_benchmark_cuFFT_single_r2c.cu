//general parts
#include <stdio.h>
#include <vector>
#include <memory>
#include <string.h>
#include <chrono>
#include <thread>
#include <iostream>
#ifndef __STDC_FORMAT_MACROS
#define __STDC_FORMAT_MACROS
#endif
#include <inttypes.h>

//CUDA parts
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define GROUP 1


void sample_6_benchmark_cuFFT_single_r2c(bool file_output, FILE* output)
{
	if (file_output)
		fprintf(output, "6 - cuFFT FFT + iFFT R2C/C2R multidimensional benchmark in single precision\n");
	printf("6 - cuFFT FFT + iFFT R2C/C2R multidimensional benchmark in single precision\n");
	const int num_benchmark_samples = 24;
	const int num_runs = 3;
	//printf("First %" PRIu64 " runs are a warmup\n", num_runs);
	uint64_t benchmark_dimensions[num_benchmark_samples][4] = { {1024, 1024, 1, 2}, {64, 64, 1, 2}, {256, 256, 1, 2}, {1024, 256, 1, 2}, {512, 512, 1, 2}, {1024, 1024, 1, 2},  {4096, 256, 1, 2}, {2048, 1024, 1, 2},{4096, 2048, 1, 2}, {4096, 4096, 1, 2}, {720, 480, 1, 2},{1280, 720, 1, 2},{1920, 1080, 1, 2}, {2560, 1440, 1, 2},{3840, 2160, 1, 2},
																{32, 32, 32, 3}, {64, 64, 64, 3}, {256, 256, 32, 3},  {1024, 256, 32, 3},  {256, 256, 256, 3}, {2048, 1024, 8, 3},  {512, 512, 128, 3}, {2048, 256, 256, 3}, {4096, 512, 8, 3} };
	
	double benchmark_result[2] = { 0,0 };//averaged result = sum(system_size/iteration_time)/num_benchmark_samples
	hipfftReal* inputC = (hipfftReal*)malloc((uint64_t)sizeof(hipfftReal)*pow(2, 27));
	for (uint64_t i = 0; i < pow(2, 27); i++) {
		inputC[i] = 2 * ((float)rand()) / RAND_MAX - 1.0;
	}
	for (int n = 0; n < num_benchmark_samples; n++) {
		double run_time[num_runs][2];
		for (int r = 0; r < num_runs; r++) {
			hipfftHandle planR2C;
			hipfftHandle planC2R;
			hipfftReal* dataR;
			hipfftComplex* dataC;

			uint64_t dims[3] = { benchmark_dimensions[n][0] , benchmark_dimensions[n][1] ,benchmark_dimensions[n][2] };

			hipMalloc((void**)&dataR, sizeof(hipfftComplex) * (dims[0] / 2 + 1) * dims[1] * dims[2]);
			hipMalloc((void**)&dataC, sizeof(hipfftComplex) * (dims[0] / 2 + 1) * dims[1] * dims[2]);

			hipMemcpy(dataR, inputC, sizeof(hipfftReal) * dims[0] * dims[1] * dims[2], hipMemcpyHostToDevice);
			if (hipGetLastError() != hipSuccess) {
				fprintf(stderr, "Cuda error: Failed to allocate\n");
				return;
			}
			switch (benchmark_dimensions[n][3]) {
			case 1:
				hipfftPlan1d(&planR2C, dims[0], HIPFFT_R2C, 1);
				hipfftPlan1d(&planC2R, dims[0], HIPFFT_C2R, 1);
				break;
			case 2:
				hipfftPlan2d(&planR2C, dims[1], dims[0], HIPFFT_R2C);
				hipfftPlan2d(&planC2R, dims[1], dims[0], HIPFFT_C2R);
				break;
			case 3:
				hipfftPlan3d(&planR2C, dims[2], dims[1], dims[0], HIPFFT_R2C);
				hipfftPlan3d(&planC2R, dims[2], dims[1], dims[0], HIPFFT_C2R);
				break;
			}

			float totTime = 0;
			uint64_t cuBufferSize = sizeof(float) * 2 * (dims[0]/2+1) * dims[1] * dims[2];
			uint64_t num_iter = ((4096 * 1024.0 * 1024.0) / cuBufferSize > 1000) ? 1000 : (4096 * 1024.0 * 1024.0) / cuBufferSize;
			if (num_iter == 0) num_iter = 1;
			std::chrono::steady_clock::time_point timeSubmit = std::chrono::steady_clock::now();
			for (int i = 0; i < num_iter; i++) {

				hipfftExecR2C(planR2C, dataR, dataC);
				hipfftExecC2R(planC2R, dataC, dataR);
			}
			hipDeviceSynchronize();
			std::chrono::steady_clock::time_point timeEnd = std::chrono::steady_clock::now();
			totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001) / num_iter;
			run_time[r][0] = totTime;
			if (n > 0) {
				if (r == num_runs - 1) {
					double std_error = 0;
					double avg_time = 0;
					for (uint64_t t = 0; t < num_runs; t++) {
						avg_time += run_time[t][0];
					}
					avg_time /= num_runs;
					for (uint64_t t = 0; t < num_runs; t++) {
						std_error += (run_time[t][0] - avg_time) * (run_time[t][0] - avg_time);
					}
					std_error = sqrt(std_error / num_runs);
					if (file_output)
						fprintf(output, "cuFFT System: %" PRIu64 "x%" PRIu64 "x%" PRIu64 " Buffer: %" PRIu64 " MB avg_time_per_step: %0.3f ms std_error: %0.3f num_iter: %" PRIu64 " benchmark: %" PRIu64 "\n", benchmark_dimensions[n][0], benchmark_dimensions[n][1], benchmark_dimensions[n][2], cuBufferSize / 1024 / 1024, avg_time, std_error, num_iter, (uint64_t)(((double)cuBufferSize / 1024) / avg_time));

					printf("cuFFT System: %" PRIu64 "x%" PRIu64 "x%" PRIu64 " Buffer: %" PRIu64 " MB avg_time_per_step: %0.3f ms std_error: %0.3f num_iter: %" PRIu64 " benchmark: %" PRIu64 "\n", benchmark_dimensions[n][0], benchmark_dimensions[n][1], benchmark_dimensions[n][2], cuBufferSize / 1024 / 1024, avg_time, std_error, num_iter, (uint64_t)(((double)cuBufferSize / 1024) / avg_time));
					benchmark_result[0] += ((double)cuBufferSize / 1024) / avg_time;
				}

			}
			hipfftDestroy(planR2C);
			hipfftDestroy(planC2R);
			hipFree(dataC);
			hipFree(dataR);
			hipDeviceSynchronize();
			//cufftComplex* output_cuFFT = (cufftComplex*)(malloc(sizeof(cufftComplex) * dims[0] * dims[1] * dims[2]));
			//cudaMemcpy(output_cuFFT, dataC, sizeof(cufftComplex) * dims[0] * dims[1] * dims[2], cudaMemcpyDeviceToHost);
			//cudaDeviceSynchronize();
			

		}
	}
	free(inputC);
	benchmark_result[0] /= (num_benchmark_samples - 1);
	if (file_output)
		fprintf(output, "Benchmark score cuFFT: %" PRIu64 "\n", (uint64_t)(benchmark_result[0]));
	printf("Benchmark score cuFFT: %" PRIu64 "\n", (uint64_t)(benchmark_result[0]));

}
