//general parts
#include <stdio.h>
#include <vector>
#include <memory>
#include <string.h>
#include <chrono>
#include <thread>
#include <iostream>
#ifndef __STDC_FORMAT_MACROS
#define __STDC_FORMAT_MACROS
#endif
#include <inttypes.h>

//CUDA parts
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>
#include <hip/hip_fp16.h>
#include <hipfft/hipfftXt.h>

void launch_precision_cuFFT_half(void* inputC, void* output_cuFFT, int device_id, uint64_t* dims)
{
	hipSetDevice(device_id);
	hipfftHandle planHalf;
	half2* dataC;
	hipMalloc((void**)&dataC, sizeof(half2) * dims[0] * dims[1] * dims[2]);
	hipMemcpy(dataC, inputC, sizeof(half2) * dims[0] * dims[1] * dims[2], hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return;
	}
	uint64_t sizeCUDA;
	hipfftResult res = hipfftCreate(&planHalf);
	size_t ws = 0;
	long long local_dims[3];
	switch (dims[3]) {
	case 1:
		local_dims[0] = (long long)dims[0];
		local_dims[1] = (long long)dims[1];
		local_dims[2] = (long long)dims[2];
		break;
	case 2:
		local_dims[0] = (long long)dims[1];
		local_dims[1] = (long long)dims[0];
		local_dims[2] = (long long)dims[2];
		break;
	case 3:
		local_dims[0] = (long long)dims[2];
		local_dims[1] = (long long)dims[1];
		local_dims[2] = (long long)dims[0];
		break;
	}
	res = hipfftXtMakePlanMany(
		planHalf, dims[3], local_dims, NULL, 1, 1, HIP_C_16F,
		NULL, 1, 1, HIP_C_16F, 1, &ws, HIP_C_16F);

	for (int i = 0; i < 1; i++) {
		res = hipfftXtExec(planHalf, dataC, dataC, -1);
	}
	hipDeviceSynchronize();
	hipMemcpy(output_cuFFT, dataC, sizeof(half2) * dims[0] * dims[1] * dims[2], hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipfftDestroy(planHalf);
	hipFree(dataC);
}
